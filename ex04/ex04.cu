#include "hip/hip_runtime.h"
#include <iostream>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 0
__global__
void saxpy(unsigned int n, double a, double *x, double *y)
{
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  hipSetDevice(MYDEVICE);

  // 1<<N is the equivalent to 2^N
  unsigned int N = 20 * (1 << 20);
  double *x, *y, *d_x, *d_y;
  x = (double*)malloc(N*sizeof(double));
  y = (double*)malloc(N*sizeof(double));

  hipMalloc(&d_x, N*sizeof(double)); 
  hipMalloc(&d_y, N*sizeof(double));

  for (unsigned int i = 0; i < N; i++) {
    x[i] = 1.0;
    y[i] = 2.0;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(double), hipMemcpyHostToDevice);

  hipEventRecord(start);

  saxpy<<<(N+511)/512, 512>>>(N, 2.0, d_x, d_y);

  hipEventRecord(stop);

  hipMemcpy(y, d_y, N*sizeof(double), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);


  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  double maxError = 0.;
  for (unsigned int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i]-4.0));
  }
  
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  
  std::cout << "ElapsedTime: " << milliseconds << std::endl;
  std::cout << "maxError: " << maxError << std::endl;

}



