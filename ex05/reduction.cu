#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <numeric>
#include <iostream>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

double random_double(void)
{

  return static_cast<double>(rand()) / RAND_MAX;
}


// Part 1 of 6: implement the kernel
__global__ void block_sum(const double *input,
                          double *per_block_results,
                          const size_t n)
{
  //fill me
  __shared__ double sdata[];
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n){
    sdata[threadIdx.x] = input[i];
    atomicAdd(per_block_results[blockIdx.x], sdata[threadIdx.x]);
  }

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(void)
{
  
  constexpr int blockDim = 1<<9;
  
  // create array of 256ki elements
  const int num_elements = 1<<18;
  srand(time(NULL));
  // generate random input on the host
  std::vector<double> h_input(num_elements);
  for(int i = 0; i < h_input.size(); ++i)
  {
    h_input[i] = random_double();
  }

  const double host_result = std::accumulate(h_input.begin(), h_input.end(), 0.0f);
  std::cerr << "Host sum: " << host_result << std::endl;

  //Part 1 of 6: move input to device memory
  double *d_input = 0;
  hipMalloc((void**)&d_input, num_elements * sizeof(double) );
  hipMemcpy(d_input, h_input, num_elements * sizeof(double), hipMemcpyHostToDevice);
  
  // Part 1 of 6: allocate the partial sums: How much space does it need?
  double *d_partial_sums_and_total = 0;
  hipMalloc((void**)&d_partial_sums_and_total, num_elements / blockDim * sizeof(double) );
  
  // Part 1 of 6: launch one kernel to compute, per-block, a partial sum. How much shared memory does it need?
  block_sum<<<num_elements / blockDim, blockDim, blockDim>>>(d_input, d_partial_sums_and_total, num_elements);

    // Part 1 of 6: copy the result back to the host
  double *d_result = 0;
  double device_result = 0;
  hipMalloc((void**)&device_result, 1 * sizeof(double));
  
  // Part 1 of 6: compute the sum of the partial sums
  block_sum<<<1, blockDim, blockDim>>>(d_partial_sums_and_total, device_result, num_elements / blockDim);
  hipMemcpy(&device_result, d_result, 1 * sizeof(double), hipMemcpyDeviceToHost);


  std::cout << "Device sum: " << device_result << std::endl;

  // Part 1 of 6: deallocate device memory

  hipFree(d_input);
  hipFree(d_partial_sums_and_total);
  hipFree(d_result);

  return 0;
}
